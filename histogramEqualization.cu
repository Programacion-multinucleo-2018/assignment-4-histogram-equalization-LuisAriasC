#include "hip/hip_runtime.h"
/*
  Author: Luis Carlos Arias Camacho
  Student ID: A01364808
  ASSIGNMENT 4
*/

#include <iostream>
#include <cstdio>
#include <cmath>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include "common.h"
#include <hip/hip_runtime.h>
#include <chrono>

#define img_dest "Images/"
#define default_image "dog.jpeg"
#define C_SIZE 256

using namespace std;

//This function is used to equalize an histogram
  // src_histogram - imput histogram as a one dimentional array of ints
  // eq_histogram - output normalized histogram as a one dimentional array of ints
  // size - size of the histograms
void normalize(int * src_histogram, int * eq_histogram, int size){
    int step = size / C_SIZE;
    int sum = 0;

    for(int i=0; i < C_SIZE; i++){
        sum += src_histogram[i];
        eq_histogram[i] = sum / step;
    }
}



// Histogram equalization on cpu
  // imput - input image
  //output - output image
  //imageName - path to achieve the image
void equalizer_cpu(const cv::Mat &input, cv::Mat &output, string imageName){

  int width = input.cols;
  int height = input.rows;
  int size_ = width * height;

  //Histogram
  int histo[C_SIZE]{};

  //Fill histogram
  for (int i = 0; i < size_; i++)
    histo[input.ptr()[i]]++;

  //Normalize histogram
  int step = size_ / C_SIZE;
  int sum = 0;
  int n_histo[C_SIZE]{};
  for(int i=0; i < C_SIZE; i++){
      sum += histo[i];
      n_histo[i] = sum / step;
  }

  //Write image with normalized histogram on output
  for (int i = 0; i < size_; i++)
    output.ptr()[i] = n_histo[input.ptr()[i]];

  //Save the image
  cv::imwrite("Images/eq_cpu_" + imageName , output);
}



//This function converts a colored imege to a grayscale image
  // input - input image one dimensional array
  // ouput - output image one dimensional array
  // width, height - width and height of the images
  // colorWidthStep - number of color bytes (cols * colors)
  // grayWidthStep - number of gray bytes
__global__ void bgr_to_gray_kernel(unsigned char* input, unsigned char* output, int width, int height, int colorWidthStep, int grayWidthStep){
	// 2D Index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	if ((xIndex < width) && (yIndex < height)){
		const int color_tid = yIndex * colorWidthStep + (3 * xIndex);
		const int gray_tid = yIndex * grayWidthStep + xIndex;
		const unsigned char blue = input[color_tid];
		const unsigned char green = input[color_tid + 1];
		const unsigned char red = input[color_tid + 2];
		const float gray = red * 0.3f + green * 0.59f + blue * 0.11f;
		output[gray_tid] = static_cast<unsigned char>(gray);
	}
}



// Get histogram with gpu and atomic operations
  //output - output image int array
  //histo - histogram of the images as an array
  // width, height - width and height of the images
  // grayWidthStep - number of gray bytes
__global__ void get_histogram_kernel(unsigned char* output, int* histo,int width, int height, int grayWidthStep){

	// 2D Index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	if ((xIndex < width) && (yIndex < height)){
    const int tid = yIndex * grayWidthStep + xIndex;
    atomicAdd(&histo[(int)output[tid]], 1);
    __syncthreads();
	}
}


// Histogram equalization on gpu
  // imput - input image
  //output - output image
  //hist - input image histogram
  // width, height - width and height of the images
  // grayWidthStep - number of gray bytes
__global__ void equalizer_kernel(unsigned char* input, unsigned char* output, int * hist, int width, int height, int grayWidthStep){

  //Initialize shared memory for block
  __shared__ int hist_s[256];

    //2D Index of current thread
	unsigned int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int yIndex = threadIdx.y + blockIdx.y * blockDim.y;

  //Index in shared memory
  unsigned int sxy = threadIdx.y * blockDim.x + threadIdx.x;
  //Thread ID
  const int tid  = yIndex * grayWidthStep + xIndex;

  //Fill in shared memory histogram
  if (sxy < 256){
    hist_s[sxy] = 0;
    hist_s[sxy] = hist[sxy];
  }
  __syncthreads();

  //Generate output image
  if((xIndex < width) && (yIndex < height))
      output[tid] = hist_s[input[tid]];
}


//Call this function to run the image equalization
  // input - input image
  // output - black & white output image
  // eq_output - equalized output image
  // imageName - path to reach the input image
void histogram_equalization(const cv::Mat& input, cv::Mat& output, cv::Mat& eq_output, string imageName){

  //Get size of the image
	size_t colorBytes = input.step * input.rows;
	size_t grayBytes = output.step * output.rows;
  int imSize = input.cols * input.rows;

  //Set device and cpu image arrays and histograms
	unsigned char *d_input, *d_output, *de_output;
  int * d_histogram, * df_histogram;
  int * histogram = (int *)malloc(C_SIZE * sizeof(int));
  int * f_histogram = (int *)malloc(C_SIZE * sizeof(int));
  for (int i = 0; i < C_SIZE; i++)
    f_histogram[i] = histogram[i] = 0;

	// Allocate device memory
	SAFE_CALL(hipMalloc<unsigned char>(&d_input, colorBytes), "CUDA Malloc Failed");
	SAFE_CALL(hipMalloc<unsigned char>(&d_output, grayBytes), "CUDA Malloc Failed");
  SAFE_CALL(hipMalloc<unsigned char>(&de_output, grayBytes), "CUDA Malloc Failed");
  SAFE_CALL(hipMalloc<int>(&d_histogram, C_SIZE * sizeof(int)), "CUDA Malloc Failed");
  SAFE_CALL(hipMalloc<int>(&df_histogram, C_SIZE * sizeof(int)), "CUDA Malloc Failed");

	// Copy data from OpenCV input image to device memory
	SAFE_CALL(hipMemcpy(d_input, input.ptr(), colorBytes, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");
  SAFE_CALL(hipMemset(d_histogram, 0, C_SIZE * sizeof(int)), "Error setting d_MatC to 0");

  const dim3 block(16, 16);
	const dim3 grid((int)ceil((float)input.cols / block.x), (int)ceil((float)input.rows/ block.y));

	// Launch the color conversion kernel
  printf("Converting image to black & white\n");
	bgr_to_gray_kernel <<<grid, block >>>(d_input, d_output, input.cols, input.rows, static_cast<int>(input.step), static_cast<int>(output.step));
  // Synchronize to check for any kernel launch errors
	SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");
  SAFE_CALL(hipMemcpy(output.ptr(), d_output, grayBytes, hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");
  //Write the black & white image
  cv::imwrite("Images/bw_" + imageName , output);

  // Launch equalization on cpu
  printf("Equalization on cpu.\n");
  float cpuTime = 0.0;
  auto start_cpu =  chrono::high_resolution_clock::now();
  equalizer_cpu(output, eq_output, imageName);
  auto end_cpu =  chrono::high_resolution_clock::now();
  chrono::duration<float, std::milli> duration_ms = end_cpu - start_cpu;
  cpuTime = duration_ms.count();

  // Set the eq_output image to 0 in order to reuse it in gpu
  memset(eq_output.ptr(), 0, colorBytes);

  //Launch histogram calculation on cpu
  printf("Equalization on gpu.\n");
  float gpuTime = 0.0;
  auto start_gpu =  chrono::high_resolution_clock::now();
  get_histogram_kernel<<<grid, block >>>(d_output, d_histogram, input.cols, input.rows, static_cast<int>(output.step));
  SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");
  auto end_gpu =  chrono::high_resolution_clock::now();
  chrono::duration<float, std::milli> gpu_duration_ms = end_gpu - start_gpu;
  gpuTime += gpu_duration_ms.count();
  // SAFE_CALL kernel error
  SAFE_CALL(hipGetLastError(), "Error with last error");

  // Copy device histogram to host histogram
  SAFE_CALL(hipMemcpy(histogram, d_histogram, C_SIZE * sizeof(int), hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");
  //Normalize histogram
  normalize(histogram, f_histogram, imSize);

  //Copy normalized histogram to device normalized histogram
  SAFE_CALL(hipMemcpy(df_histogram, f_histogram, C_SIZE * sizeof(int), hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");

  //Set output image with normalized histogram
  start_gpu =  chrono::high_resolution_clock::now();
  equalizer_kernel<<<grid, block >>>(d_output, de_output, df_histogram, input.cols, input.rows, static_cast<int>(output.step));
  SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");
  end_gpu =  chrono::high_resolution_clock::now();
  gpu_duration_ms = end_gpu - start_gpu;
  gpuTime += gpu_duration_ms.count();

  //Write the black & white equalized image
  SAFE_CALL(hipMemcpy(eq_output.ptr(), de_output, grayBytes, hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");
  cv::imwrite("Images/eq_gpu_" + imageName , eq_output);

  printf("Time in CPU: %f\n", cpuTime);
  printf("Time in GPU: %f\n", gpuTime);
  printf("Speedup: %f\n", cpuTime / gpuTime );

	// Free the device memory
	SAFE_CALL(hipFree(d_input), "CUDA Free Failed");
	SAFE_CALL(hipFree(d_output), "CUDA Free Failed");
  SAFE_CALL(hipFree(de_output), "CUDA Free Failed");
  SAFE_CALL(hipFree(d_histogram), "CUDA Free Failed");
  SAFE_CALL(hipFree(df_histogram), "CUDA Free Failed");

  //Free the host memory
  free(histogram);
  free(f_histogram);

  // Reset device
  SAFE_CALL(hipDeviceReset(), "Error reseting");
}

int main(int argc, char *argv[]){

	string inputImage;

	if(argc < 2)
		inputImage = default_image;
  	else
  		inputImage = argv[1];

	// Read input image from the disk
	cv::Mat input = cv::imread(img_dest + inputImage, CV_LOAD_IMAGE_COLOR);

	if (input.empty()){
		cout << "Image Not Found!" << std::endl;
		cin.get();
		return -1;
	}

	//Create output image
	cv::Mat output(input.rows, input.cols, CV_8UC1);
  	//Create equalized output image
  	cv::Mat eq_output(input.rows, input.cols, CV_8UC1);

	//Convert image to gray and equalize
	histogram_equalization(input, output, eq_output, inputImage);

	//Allow the windows to resize
	namedWindow("Input", cv::WINDOW_NORMAL);
	namedWindow("Blac&WhiteInput", cv::WINDOW_NORMAL);
	namedWindow("Output", cv::WINDOW_NORMAL);

	//Show the input and output
	imshow("Input", input);
	imshow("Blac&WhiteInput", output);
  	imshow("Output", eq_output);
	//Wait for key press
	cv::waitKey();
	return 0;
}
